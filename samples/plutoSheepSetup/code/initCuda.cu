#include "hip/hip_runtime.h"
extern "C" {

#include "pluto.h"
#include "plutoCuda.cuh"

#define PI 3.141592654

/* ************************************************************************** */
__device__ void cudaUserDefBoundary (int side, uint3 dataIdx, dim3 dataDim)
/*!
 *	Assign user-defined boundary conditions.
 *
 * \param [in] side	  specifies the boundary side where ghost zones need
 *			  to be filled. It can assume the following
 *			  pre-definite values: X1_BEG, X1_END,
 *					   X2_BEG, X2_END,
 *					   X3_BEG, X3_END.
 * \param [in] dataIdx
 * \param [in] dataDim
 *
 * The primitive variables of the ghost cells can be set the following way:
 *	cudaV.rho[gpu1D(dataIdx, dataDim)] = ...
 *	cudaV.vx1[gpu1D(dataIdx, dataDim)] = ...
 *	cudaV.vx2[gpu1D(dataIdx, dataDim)] = ... (only if COMPONENTS > 1)
 *	cudaV.vx3[gpu1D(dataIdx, dataDim)] = ... (only if COMPONENTS > 2)
 *	cudaV.prs[gpu1D(dataIdx, dataDim)] = ...
 *
 * The coordinates of the cell can be accessed the following way:
 *	x1 coordinate: cudaGrid.x1[dataIdx.x];
 *	x2 coordinate: cudaGrid.x2[dataIdx.y];
 *	x3 coordinate: cudaGrid.x3[dataIdx.z];
 *
 **************************************************************************** */
{
	int idx = gpu1D(dataIdx, dataDim);

	/* Density floor */
	if (cudaV.rho[idx] < cudaSmallDensity)
    {
		cudaV.rho[idx] = cudaSmallDensity;
	}

	/* Wave damping in radial direction */
	if (side == 0)
	{
		real r = cudaGrid.xgc1[dataIdx.x];
		real th = cudaGrid.xgc2[dataIdx.y];

		real beta = abs(th - 0.5*CONST_PI);

		real r_wd_in = cudaInputParam[WDRIn];
		real r_wd_out = cudaInputParam[WDROut];

		int bndIdx = 0;
		// access r_min
		real r_min = cudaGrid.xgc1[cudaIBEG];
		// access r_max
		real r_max = cudaGrid.xgc1[cudaIEND];
		// access theta_min
		real beta_max = 0.5*CONST_PI - cudaGrid.xgc2[cudaJBEG];

		real delta = cudaInputParam[WDThetaBegRel]*beta_max;

		if ( r <= r_wd_in || r >= r_wd_out || beta >= delta ) {
			/* Wave damping */


			real tau_r = 1.0;
			real ramp_r = 1.0;

			real tau_th = 1.0;
			real ramp_th = 1.0;

			real tau = 1.0;
			real ramp = 1.0;

			/* Select the highest damping factor.
			   This avoids using an amplitude higher than 1. */
			if ( r <= r_wd_in ) {
				/* inner wave killing zone */
				tau_r = sqrt(r_min*r_min*r_min);
				ramp_r = (r - r_wd_in)*(r - r_wd_in) / ( (r_wd_in - r_min)*(r_wd_in - r_min) );
			}

			if ( r >= r_wd_out ) {
				/* outer wave killing zone */
				tau_r = sqrt(r_max*r_max*r_max);
				ramp_r = (r - r_wd_out)*(r - r_wd_out) / ( (r_max - r_wd_out)*(r_max - r_wd_out) );
			}

			if ( beta >= delta ) {
				tau_th = sqrt(r*r*r);
				ramp_th = (beta-delta)*(beta-delta) / ( (beta_max - delta)*(beta_max -delta) );
			}

			if ( ramp_th >= ramp_r ) {
				ramp = ramp_th;
				tau = tau_th;
			} else {
				ramp = ramp_r;
				tau = tau_r;
			}

			/* inside a wave killing zone */
			real lambda = cuda_dt/tau*ramp;
			/* Calculate equilibrium values */
			real h = cudaInputParam[AspectRatio];
			real rho_c = cudaInputParam[SigmaRef]/(sqrt(2.0*PI)*h);
			real sin_th = sin(th);
			real R = r*sin_th;
			real H = R*h;
			real z = r*cos(th);
			real R_m32 = 1.0/(R*sqrt(R));
			real OmegaK = 2.0*PI*R_m32;

			/* Equilibrium values as in init.c */
			real rho = rho_c*R_m32*exp((sin_th-1)/(h*h));
			real v_phi = R*OmegaK*sqrt(-2.5*h*h+sin_th);

			cudaV.vc[RHO][idx] += - (cudaV.vc[RHO][idx] - rho)*lambda;
			cudaV.vc[VX3][idx] += - (cudaV.vc[VX3][idx] - v_phi)*lambda;
			/* Damp radial and vertical velocity to 0. */
			cudaV.vc[VX2][idx] += - (cudaV.vc[VX2][idx])*lambda;
			cudaV.vc[VX1][idx] += - (cudaV.vc[VX1][idx])*lambda;

			if (cudaV.rho[idx] < cudaSmallDensity)
			{
				cudaV.rho[idx] = cudaSmallDensity;
			}

		}
	}

}

#if BODY_FORCE != NO
/* ********************************************************************* */
__device__ real cudaBodyForceVector(int dir, uint3 dataIdx, dim3 dataDim)
/*!
 * Returns the component of the acceleration vector in a given direction.
 *
 * \param [in] dir	specifies the component of the acceleration vector.
 * \param [in] dataIdx
 * \param [in] dataDim
 *
 * The coordinates of the cell can be accessed the following way:
 *	x1 coordinate: cudaGrid.x1[dataIdx.x];
 *	x2 coordinate: cudaGrid.x2[dataIdx.y];
 *	x3 coordinate: cudaGrid.x3[dataIdx.z];
 *
 * The cell-centered primitive variables of the cell can be accessed
 * the following way:
 *	rho: cudaV.rho[gpu1D(dataIdx, dataDim)]
 *	vx1: cudaV.vx1[gpu1D(dataIdx, dataDim)]
 *	vx2: cudaV.vx2[gpu1D(dataIdx, dataDim)] (only if COMPONENTS > 1)
 *	vx3: cudaV.vx3[gpu1D(dataIdx, dataDim)] (only if COMPONENTS > 2)
 *	prs: cudaV.prs[gpu1D(dataIdx, dataDim)]
 *
 *********************************************************************** */
{
	return 0.0;
}

/* ********************************************************************* */
__device__ real cudaBodyForcePotential(real x1, real x2, real x3)
/*!
 * Returns the gravitational potential as a function of the coordinates.
 *
 * \param [in] x1  position in the 1st coordinate direction \f$x_1\f$
 * \param [in] x2  position in the 2nd coordinate direction \f$x_2\f$
 * \param [in] x3  position in the 3rd coordinate direction \f$x_3\f$
 *
 * \return The body force potential \f$ \Phi(x_1,x_2,x_3) \f$.
 *
 *********************************************************************** */
{
	real x, y, z;
	real x_p, y_p, z_p;
	real x_s, y_s, z_s;
	real r_hill, r_sm, r_sm_ip4, r_sm_ip3, r_sm_ip1;
	real mp ,mu, phi, phiplanet;
	real s_star, s_p;

	mu = cudaInputParam[Mplanet];

	/* get planet and star position */
	x_p = cpuCalcParams[P_POS_X];
	y_p = cpuCalcParams[P_POS_Y];
	z_p = cpuCalcParams[P_POS_Z];
	x_s = cpuCalcParams[S_POS_X];
	y_s = cpuCalcParams[S_POS_Y];
	z_s = cpuCalcParams[S_POS_Z];

	/* calculate position of cell center */
	x = x1*sin(x2)*cos(x3);
	y = x1*sin(x2)*sin(x3);
	z = x1*cos(x2);

	/* calculate distance of cell center to star and to planet */
	s_star = sqrt( (x_s-x)*(x_s-x) + (y_s-y)*(y_s-y) + (z_s-z)*(z_s-z) );
	s_p =	 sqrt( (x_p-x)*(x_p-x) + (y_p-y)*(y_p-y) + (z_p-z)*(z_p-z) );

	#if USE_PLANET_POTENTIAL == YES
		/* Smoothing length = parameter*R_hill */
		r_hill = pow(mu/3.0, 1.0/3.0);
		r_sm = r_hill*cudaInputParam[Smoothing];
		/* inverse powers of the smoothing length */
		r_sm_ip1 = 1.0/r_sm;
		r_sm_ip3 = 1.0/(r_sm*r_sm*r_sm);
		r_sm_ip4 = 1.0/(r_sm*r_sm*r_sm*r_sm);
		/* calculate planet potential */
		if ( s_p  > r_sm ) phiplanet = mu/s_p;
		else phiplanet = mu*(s_p*s_p*s_p*r_sm_ip4 - 2.*s_p*s_p*r_sm_ip3 + 2.*r_sm_ip1);
	#else
		phiplanet = 0.0;
	#endif

	/* calculate gravitational potential of star and planet */
	phi	 = - 4.0*PI*PI*(1.0/s_star + phiplanet);

	return phi;
}
#endif

#if VISCOSITY != NO
__device__ void cudaViscNu(real *v, double x1, double x2, double x3,
					double *nu1, double *nu2)
{
	double Omega_K, R, H,alpha;
	R = x1*sin(x2);
	H = cudaInputParam[AspectRatio]*R;
	alpha = cudaInputParam[ViscosityAlpha];
	Omega_K = 2.0*PI/(R*sqrt(R));
	/* Calculate alpha viscosity and multiply by rho b/c pluto wants it. */
	*nu1 = alpha*H*H*Omega_K*v[RHO];
	*nu2 = 0.0;
}
#endif

#if USE_CUDA_REDUCTION == YES
__device__ void calcReductionValues(real *values, uint3 dataIdx, dim3 dataDim)
{

}
#endif

#if USE_CUDA_ANALYSIS == YES
__device__ void calcAnalysisValues(real *analysisValues, uint3 dataIdx, dim3 dataDim)
{
	int idx = gpu1D(dataIdx, dataDim);

	real x_p = cpuCalcParams[P_POS_X];
	real y_p = cpuCalcParams[P_POS_Y];
	real z_p = cpuCalcParams[P_POS_Z];

	real r, phi, th;
	real sin_th, cos_th, sin_phi, cos_phi;
	real vr, vth, vphi;
	real dV, dm, rho;
	real r_hill;
	real s, s_x, s_y, s_z;
	real force, cutoff;
	real mu, mp, b;
	real jx, jy, jz;

	/* get planet mass and calculate hill radius */
	mu = cudaInputParam[Mplanet];
	mp = cudaInputParam[Mplanet];
	r_hill = pow(mu/3.0, 1.0/3.0);


	/* get position of cell in spherical coordinates */
	r = cudaGrid.xgc1[dataIdx.x];
	th = cudaGrid.xgc2[dataIdx.y];
	phi = cudaGrid.xgc3[dataIdx.z];
	/* Get sines and cosines of th and phi */
	sin_th = sin(th);
	cos_th = cos(th);
	sin_phi = sin(phi);
	cos_phi = cos(phi);

	/* get velocities of cells in spherical coordinates */
	vr = cudaV.vc[VX1][idx];
	vth = cudaV.vc[VX2][idx];
	vphi = cudaV.vc[VX3][idx];

	/* Get cell colume and calculate cell mass. */
	rho = cudaV.rho[idx];
	dV =  cudaGrid.dVx1[dataIdx.x]
	 *cudaGrid.dVx2[dataIdx.y]
	 *cudaGrid.dVx3[dataIdx.z];
	dm = rho * dV;

	b = cudaInputParam[ForceCutoff];
	/*************** end of gpu specific code ****************/

	/* Calculate vector and distance from cell center to planet. */
	s_x = x_p - r*cos_phi*sin_th;
	s_y = y_p - r*sin_phi*sin_th;
	s_z = z_p - r*cos_th;
	s = sqrt(s_x*s_x + s_y*s_y + s_z*s_z);


	/* Fermi type cutoff as in Kley et. al. (2009).*/
	cutoff = 1.0/ (1.0 + exp( -10.0*(s/r_hill - b)/b ) );

	/* Calculate force and store it to analysis value array. */
	force = 4.0*PI*PI/(s*s*s)*cutoff;

	analysisValues[AN_F] = - dm*mp*s_x*force;
	analysisValues[AN_F + 1] = - dm*mp*s_y*force;
	analysisValues[AN_F + 2] = - dm*mp*s_z*force;

	/* Calculate total mass of disk.
	   Write to value array, remember to just set the variable insted of +=.
	   the code takes care of the reduction. */
	analysisValues[AN_M_DISK] = dm;

	/* Min and max density */
	analysisValues[AN_RHO_MIN] = rho;
	analysisValues[AN_RHO_MAX] = rho;

	/* Calculate kinetic energy of cell and the contribution from each direction. */
	analysisValues[AN_KE_R] = 0.5*dm*vr*vr;
	analysisValues[AN_KE_TH] = 0.5*dm*vth*vth;
	analysisValues[AN_KE_PHI] = 0.5*dm*vphi*vphi;


	/* Calculate the contribution of each cell to the angular momentum of the disk. */
	jx = dm*r*( -vth*sin_phi - vphi*cos_th*cos_phi );
	jy = dm*r*( vth*cos_phi	 - vphi*cos_th*sin_phi );
	jz = dm*r*( vphi*sin_th );

	analysisValues[AN_J_DISK] = jx;
	analysisValues[AN_J_DISK+1] = jy;
	analysisValues[AN_J_DISK+2] = jz;
}

#endif

#if USE_CPU_CALC_PARAMETERS == YES
void calcCpuParams(real *values)
{
	double t, Omega_P, peri, xi, inc, cos_xi, sin_xi, cos_i, sin_i, mu, r;

	inc = g_inputParam[Inclination]/180*PI;
	mu = g_inputParam[Mplanet];
	peri = g_inputParam[Pericenter];


	/* Planet position is fixed to 1 (choice of units) */
	r = 1.0;

	Omega_P = 2.0*CONST_PI;

	/* Time of actual integration step */
	if (g_intStage == 1)
	t = g_time;
	else if (g_intStage == 2)
	t = g_time + g_dt;
	else if (g_intStage == 3)
	t = g_time + 0.5*g_dt;

	xi = Omega_P*t + peri;

	cos_i = cos(inc);
	sin_i = sin(inc);
	cos_xi = cos(xi);
	sin_xi = sin(xi);

	values[P_POS_X] = 1.0/(1.0+mu)*r*cos_xi;
	values[P_POS_Y] = 1.0/(1.0+mu)*r*cos_i*sin_xi;
	values[P_POS_Z] = 1.0/(1.0+mu)*r*sin_i*sin_xi;
	values[S_POS_X] = -mu/(1.0+mu)*r*cos_xi;
	values[S_POS_Y] = -mu/(1.0+mu)*r*cos_i*sin_xi;
	values[S_POS_Z] = -mu/(1.0+mu)*r*sin_i*sin_xi;

}
#endif

} /* extern "C" */
